#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "plugin/yolov8PoseLayerPlugin/yolov8PoseLayerParameters.h"
#include "trt/trt_cuda.h"

__device__ 
float sigmoid_gpu(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__device__
float iou_gpu(
    const float xmin_A, const float ymin_A, const float xmax_A, const float ymax_A,
    const float xmin_B, const float ymin_B, const float xmax_B, const float ymax_B
) {
    float xmin = fmaxf(xmin_A, xmin_B);
    float ymin = fmaxf(ymin_A, ymin_B);
    float xmax = fminf(xmax_A, xmax_B);
    float ymax = fminf(ymax_A, ymax_B);
    float iw = fmaxf(0.0f, xmax - xmin);
    float ih = fmaxf(0.0f, ymax - ymin);
    float inter = iw * ih;
    float area1 = (xmax_A- xmin_A) * (ymax_A - ymin_A);
    float area2 = (xmax_B - xmin_B) * (ymax_B - ymin_B);
    return inter / (area1 + area2 - inter);
}

__global__
void YOLOv8PoseLayerNMS(
    YOLOv8PoseLayerParameters param,
    float* regData, float* clsData, float* psData, int headStart, int headEnd,
    float* outputRects, int* outputClasses, int* outputCount, int* outputKeep,
    int* __restrict__ numDetectionsOutput, int* __restrict__ nmsClassesOutput, 
    float* __restrict__ nmsScoresOutput, float* __restrict__ nmsBoxesOutput, 
    float* __restrict__ nmsKeyPointsOutput
) {
    int imageIdx = blockIdx.x;
    int anchorIdx = blockIdx.y * blockDim.x + threadIdx.x;
    if (imageIdx >= param.batchSize) return;
    if (anchorIdx >= param.numAnchors) return;

    int head_idx = (anchorIdx < headStart) ? 0 : (anchorIdx < headEnd) ? 1 : 2;

    int local_start = (head_idx == 1) * headStart + (head_idx == 2) * headEnd;
    
    int stride = param.minStride << head_idx;

    int h = param.inputHeight / stride;
    int w = param.inputWidth / stride;
    int local_idx = anchorIdx - local_start;
    int row = local_idx / w;
    int col = local_idx % w;

    float cx = float(col + 0.5);
    float cy = float(row + 0.5);

    float cls_max = -1;
    int cls_index = -1;
    for (int cl = 0; cl < param.numClasses; cl++) {
        float cls_val = clsData[cl * param.numAnchors + anchorIdx];
        if (cls_val > cls_max) {
            cls_max = cls_val;
            cls_index = cl;
        }
    }

    if (cls_max < param.scoreThreshold) return;

    float dx1 = regData[0 * param.numAnchors + anchorIdx];
    float dy1 = regData[1 * param.numAnchors + anchorIdx];
    float dx2 = regData[2 * param.numAnchors + anchorIdx];
    float dy2 = regData[3 * param.numAnchors + anchorIdx];

    float xmin = (cx - dx1) * stride;
    float ymin = (cy - dy1) * stride;
    float xmax = (cx + dx2) * stride;
    float ymax = (cy + dy2) * stride;
    
    xmin = fmaxf(0.0f, xmin);
    ymin = fmaxf(0.0f, ymin);
    xmax = fminf(param.inputWidth, xmax);
    ymax = fminf(param.inputHeight, ymax);

    int batch_offset = imageIdx * param.numAnchors;

    int id = atomicAdd(&outputCount[imageIdx], 1);

    int index_i = id + batch_offset;
    outputKeep[index_i] = 1;

    float* index_i_rect = outputRects + index_i * (4 + 1);
    int* index_i_class = outputClasses + index_i;
    index_i_rect[0] = xmin;
    index_i_rect[1] = ymin;
    index_i_rect[2] = xmax;
    index_i_rect[3] = ymax;
    index_i_rect[4] = cls_max;
    index_i_class[0] = cls_index;

    __syncthreads();

    for (int j = 0; j < outputCount[imageIdx]; j++) {
        int index_j = j + batch_offset;

        float* index_j_rect = outputRects + index_j * (4 + 1);
        int* index_j_class = outputClasses + index_j;

        if (index_i == index_j || outputKeep[index_j] == 0) continue;
        if (index_i_class[0] == index_j_class[0] && index_i_rect[4] < index_j_rect[4]) {
            float iou = iou_gpu(
                index_i_rect[0], index_i_rect[1], index_i_rect[2], index_i_rect[3], 
                index_j_rect[0], index_j_rect[1], index_j_rect[2], index_j_rect[3]
            );
            if (iou > param.iouThreshold) {
                outputKeep[index_i] = 0;
            }
        }
    }

    __syncthreads();

    if (outputKeep[index_i] == 1) {
        // NumDetections
        int kid = atomicAdd(&numDetectionsOutput[imageIdx], 1);

        // DetectionClasses
        nmsClassesOutput[kid + imageIdx * param.numOutputBoxes] = index_i_class[0];

        // DetectionScores
        nmsScoresOutput[kid + imageIdx * param.numOutputBoxes] = index_i_rect[4];

        // DetectionBoxes
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 0] = index_i_rect[0];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 1] = index_i_rect[1];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 2] = index_i_rect[2];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 3] = index_i_rect[3];

        // DetectionKeyPoints
        for (int k = 0; k < param.numKeypoints; k++) {
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 0] = (psData[(k * 3 + 0) * param.numAnchors + anchorIdx] * 2 + (cx - 0.5f)) * stride;
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 1] = (psData[(k * 3 + 1) * param.numAnchors + anchorIdx] * 2 + (cy - 0.5f)) * stride;
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 2] = sigmoid_gpu(psData[(k * 3 + 2) * param.numAnchors + anchorIdx]);
        }
    }
}

void YOLOv8PoseLayerLauncher(
    YOLOv8PoseLayerParameters param,
    float* regInput, float* clsInput, float* psInput,
    void* numDetectionsOutput, void* nmsClassesOutput, void* nmsScoresOutput, 
    void* nmsBoxesOutput, void* nmsKeyPointsOutput
) {
    checkCudaRuntime(hipMemset(numDetectionsOutput, 0, sizeof(int) * param.batchSize));
    checkCudaRuntime(hipMemset(nmsClassesOutput, 0, sizeof(int) * param.batchSize * param.numOutputBoxes));
    checkCudaRuntime(hipMemset(nmsScoresOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes));
    checkCudaRuntime(hipMemset(nmsBoxesOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes * 4));
    checkCudaRuntime(hipMemset(nmsKeyPointsOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes * param.numKeypoints * 3));

    float* outputRects = nullptr;
    int rects_element = (4 + 1) * param.batchSize * param.numAnchors;
    checkCudaRuntime(hipMalloc(&outputRects, rects_element * sizeof(float)));

    int* outputClasses = nullptr;
    int socres_element = param.batchSize * param.numAnchors;
    checkCudaRuntime(hipMalloc(&outputClasses, socres_element * sizeof(int)));

    int* outputCount = nullptr;
    int count_element = param.batchSize;
    checkCudaRuntime(hipMalloc(&outputCount, count_element * sizeof(int)));
    checkCudaRuntime(hipMemset(outputCount, 0, sizeof(int) * count_element));

    int* outputKeep = nullptr;
    int keep_element = param.batchSize * param.numAnchors;
    checkCudaRuntime(hipMalloc(&outputKeep, keep_element * sizeof(int)));
    checkCudaRuntime(hipMemset(outputKeep, -1, keep_element * sizeof(int)));

    int headStart = (param.inputHeight / param.minStride) * (param.inputWidth / param.minStride);
    int midMapSize = (param.inputHeight / (param.minStride << 1)) * (param.inputWidth / (param.minStride << 1));
    int headEnd = headStart + midMapSize;

    int threadSize = 256;
    dim3 block(threadSize, 1);
    dim3 grid(param.batchSize, (param.numAnchors + threadSize - 1) / threadSize);

    YOLOv8PoseLayerNMS<<<grid, block>>>(
        param,
        regInput, clsInput, psInput, headStart, headEnd,
        outputRects, outputClasses, outputCount, outputKeep,
        (int *)numDetectionsOutput, (int *)nmsClassesOutput, 
        (float *)nmsScoresOutput, (float *)nmsBoxesOutput, 
        (float *)nmsKeyPointsOutput
    );

    checkCudaRuntime(hipFree(outputKeep));
    checkCudaRuntime(hipFree(outputCount));
    checkCudaRuntime(hipFree(outputClasses));
    checkCudaRuntime(hipFree(outputRects));
}

void YOLOv8PoseLayerInference(
    YOLOv8PoseLayerParameters param,
    float* regInput, float* clsInput, float* psInput,
    int regSize, int clsSize, int psSize,
    int* numDetectionsOutput, int* nmsClassesOutput, float* nmsScoresOutput, 
    float* nmsBoxesOutput, float* nmsKeyPointsOutput
) {

    float* d_regInput = nullptr;
    float* d_clsInput = nullptr;
    float* d_psInput = nullptr;

    checkCudaRuntime(hipMalloc(&d_regInput, regSize * sizeof(float)));
    checkCudaRuntime(hipMalloc(&d_clsInput, clsSize * sizeof(float)));
    checkCudaRuntime(hipMalloc(&d_psInput, psSize * sizeof(float)));

    checkCudaRuntime(hipMemcpy(d_regInput, regInput, regSize * sizeof(float), hipMemcpyHostToDevice));
    checkCudaRuntime(hipMemcpy(d_clsInput, clsInput, clsSize * sizeof(float), hipMemcpyHostToDevice));
    checkCudaRuntime(hipMemcpy(d_psInput, psInput, psSize * sizeof(float), hipMemcpyHostToDevice));
    

    void* d_numDetectionsOutput = nullptr;
    void* d_nmsClassesOutput    = nullptr;
    void* d_nmsScoresOutput     = nullptr;
    void* d_nmsBoxesOutput      = nullptr;
    void* d_nmsKeyPointsOutput  = nullptr;

    checkCudaRuntime(hipMalloc(&d_numDetectionsOutput, sizeof(int)));
    checkCudaRuntime(hipMalloc(&d_nmsClassesOutput, sizeof(int) * param.numOutputBoxes));
    checkCudaRuntime(hipMalloc(&d_nmsScoresOutput, sizeof(float) * param.numOutputBoxes));
    checkCudaRuntime(hipMalloc(&d_nmsBoxesOutput, sizeof(float) * param.numOutputBoxes * 4));
    checkCudaRuntime(hipMalloc(&d_nmsKeyPointsOutput, sizeof(float) * param.numOutputBoxes * 3 * param.numKeypoints));

    checkCudaRuntime(hipMemset(d_numDetectionsOutput, 0, sizeof(int)));
    checkCudaRuntime(hipMemset(d_nmsClassesOutput, 0, sizeof(int) * param.numOutputBoxes));
    checkCudaRuntime(hipMemset(d_nmsScoresOutput, 0, sizeof(float) * param.numOutputBoxes));
    checkCudaRuntime(hipMemset(d_nmsBoxesOutput, 0, sizeof(float) * param.numOutputBoxes * 4));
    checkCudaRuntime(hipMemset(d_nmsKeyPointsOutput, 0, sizeof(float) * param.numOutputBoxes * 3 * param.numKeypoints));

    YOLOv8PoseLayerLauncher(
        param,
        d_regInput, d_clsInput, d_psInput,
        d_numDetectionsOutput, d_nmsClassesOutput, d_nmsScoresOutput,
        d_nmsBoxesOutput, d_nmsKeyPointsOutput
    );

    checkCudaRuntime(hipMemcpy(numDetectionsOutput, d_numDetectionsOutput, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaRuntime(hipMemcpy(nmsClassesOutput, d_nmsClassesOutput, sizeof(int) * param.numOutputBoxes, hipMemcpyDeviceToHost));
    checkCudaRuntime(hipMemcpy(nmsScoresOutput, d_nmsScoresOutput, sizeof(float) * param.numOutputBoxes, hipMemcpyDeviceToHost));
    checkCudaRuntime(hipMemcpy(nmsBoxesOutput, d_nmsBoxesOutput, sizeof(float) * param.numOutputBoxes * 4, hipMemcpyDeviceToHost));
    checkCudaRuntime(hipMemcpy(nmsKeyPointsOutput, d_nmsKeyPointsOutput, sizeof(float) * param.numOutputBoxes * 3 * param.numKeypoints, hipMemcpyDeviceToHost));

    checkCudaRuntime(hipFree(d_nmsKeyPointsOutput));
    checkCudaRuntime(hipFree(d_nmsBoxesOutput));
    checkCudaRuntime(hipFree(d_nmsScoresOutput));
    checkCudaRuntime(hipFree(d_nmsClassesOutput));
    checkCudaRuntime(hipFree(d_numDetectionsOutput));

    checkCudaRuntime(hipFree(d_psInput));
    checkCudaRuntime(hipFree(d_clsInput));
    checkCudaRuntime(hipFree(d_regInput));
}