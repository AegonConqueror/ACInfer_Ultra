#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "yolov8PoseLayerParameters.h"

__device__ 
float sigmoid_gpu(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__device__
float iou_gpu(
    const int xmin_A, const int ymin_A, const int xmax_A, const int ymax_A,
    const int xmin_B, const int ymin_B, const int xmax_B, const int ymax_B
) {
    float xmin = fmaxf(xmin_A, xmin_B);
    float ymin = fmaxf(ymin_A, ymin_B);
    float xmax = fminf(xmax_A, xmax_B);
    float ymax = fminf(ymax_A, ymax_B);
    float iw = fmaxf(0.0f, xmax - xmin);
    float ih = fmaxf(0.0f, ymax - ymin);
    float inter = iw * ih;
    float area1 = (xmax_A- xmin_A) * (ymax_A - ymin_A);
    float area2 = (xmax_B - xmin_B) * (ymax_B - ymin_B);
    return inter / (area1 + area2 - inter);
}

__global__
void YOLOv8PoseLayerNMS(
    YOLOv8PoseLayerParameters param,
    const float* reg1Data, const float* reg2Data, const float* reg3Data, 
    const float* cls1Data, const float* cls2Data, const float* cls3Data, 
    const float* ps1Data, const float* ps2Data, const float* ps3Data,
    int* outputRects, float* outputSocres, int* outputCount, int* outputKeep,
    int* __restrict__ numDetectionsOutput, int* __restrict__ nmsClassesOutput, 
    float* __restrict__ nmsScoresOutput, float* __restrict__ nmsBoxesOutput, 
    float* __restrict__ nmsKeyPointsOutput
) {
    int imageIdx = blockIdx.x;
    int anchorIdx = blockIdx.y * blockDim.x + threadIdx.x;
    if (imageIdx >= 2) return;
    if (anchorIdx >= param.numAnchors) return;

    int head_idx = 0;
    int base_tid = anchorIdx % param.numAnchors;

    if (base_tid < param.headStart)
        head_idx = 0;
    else if (base_tid < param.headEnd)
        head_idx = 1;
    else
        head_idx = 2;

    int local_start = (head_idx == 1) * param.headStart + (head_idx == 2) * param.headEnd;
    
    int stride = param.minStride << head_idx;

    int h = param.inputHeight / stride;
    int w = param.inputWidth / stride;
    int local_idx = base_tid - local_start;
    int row = local_idx / w;
    int col = local_idx % w;
    
    const float* reg = (head_idx == 0) ? reg1Data + imageIdx * param.reg1Size : ((head_idx == 1) ? reg2Data + imageIdx * param.reg2Size: reg3Data + imageIdx * param.reg3Size);
    const float* cls = (head_idx == 0) ? cls1Data + imageIdx * param.cls1Size: ((head_idx == 1) ? cls2Data + imageIdx * param.cls2Size : cls3Data + imageIdx * param.cls3Size);
    const float* ps  = (head_idx == 0) ? ps1Data + imageIdx * param.ps1Size : ((head_idx == 1) ? ps2Data + imageIdx * param.ps2Size : ps3Data + imageIdx * param.ps3Size);

    float cx = float(col + 0.5);
    float cy = float(row + 0.5);

    float cls_max = -1;
    int cls_index = -1;
    for (int cl = 0; cl < param.numClasses; cl++) {
        float cls_val = sigmoid_gpu(cls[cl * h * w + row * w + col]);
        if (cls_val > cls_max) {
            cls_max = cls_val;
            cls_index = cl;
        }
    }

    if (cls_max < param.scoreThreshold) return;

    float dx1 = reg[0 * h * w + row * w + col];
    float dy1 = reg[1 * h * w + row * w + col];
    float dx2 = reg[2 * h * w + row * w + col];
    float dy2 = reg[3 * h * w + row * w + col];

    float xmin = (cx - dx1) * stride;
    float ymin = (cy - dy1) * stride;
    float xmax = (cx + dx2) * stride;
    float ymax = (cy + dy2) * stride;
    
    xmin = fmaxf(0.0f, xmin);
    ymin = fmaxf(0.0f, ymin);
    xmax = fminf(param.inputWidth, xmax);
    ymax = fminf(param.inputHeight, ymax);

    int batch_offset = imageIdx * param.numAnchors;

    int id = atomicAdd(&outputCount[imageIdx], 1);

    int index_i = id + batch_offset;
    outputKeep[index_i] = 1;

    int* index_i_rect = outputRects + index_i * (4 + 1);
    float* index_i_score = outputSocres + index_i;
    index_i_rect[0] = int(xmin / param.inputWidth + 0.5);
    index_i_rect[1] = int(ymin / param.inputHeight + 0.5);
    index_i_rect[2] = int(xmax / param.inputWidth + 0.5);
    index_i_rect[3] = int(ymax / param.inputHeight + 0.5);
    index_i_rect[4] = cls_index;
    index_i_score[0] = cls_max;

    __syncthreads();

    for (int j = 0; j < outputCount[imageIdx]; j++) {
        int index_j = j + batch_offset;

        int* index_j_rect = outputRects + index_j * (4 + 1);
        float* index_j_score = outputSocres + index_j;

        if (index_i == index_j || outputKeep[index_j] == 0) continue;
        if (index_i_score[0] < index_j_score[0] &&  index_i_rect[4] == index_j_rect[4]) {
            float iou = iou_gpu(
                index_i_rect[0], index_i_rect[1], index_i_rect[2], index_i_rect[3], 
                index_j_rect[0], index_j_rect[1], index_j_rect[2], index_j_rect[3]
            );

            if (iou > param.iouThreshold) {
                outputKeep[index_i] = 0;
            }
        }
    }

    __syncthreads();

    if (outputKeep[index_i] == 1) {
        // NumDetections
        int kid = atomicAdd(&numDetectionsOutput[imageIdx], 1);

        // DetectionClasses
        nmsClassesOutput[kid + imageIdx * param.numOutputBoxes] = index_i_rect[4];

        // DetectionScores
        nmsScoresOutput[kid + imageIdx * param.numOutputBoxes] = index_i_score[0];

        // DetectionBoxes
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 0] = index_i_rect[0];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 1] = index_i_rect[1];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 2] = index_i_rect[2];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 3] = index_i_rect[3];

        // DetectionKeyPoints
        // float* det_keypoints_batch = det_keypoints + imageIdx * topK * 3 * num_keypoints;
        for (int k = 0; k < param.numKeypoints; k++) {
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 0] = (ps[(k * 3 + 0) * h * w + row * w + col] * 2 + (cx - 0.5f)) * stride  / param.inputWidth;
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 1] = (ps[(k * 3 + 1) * h * w + row * w + col] * 2 + (cy - 0.5f)) * stride  / param.inputHeight;
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 2] = sigmoid_gpu(ps[(k * 3 + 2) * h * w + row * w + col]);
        }
    }
}

template <typename T>
T* YOLOv8PoseLayerWorkspace(void* workspace, size_t& offset, size_t elements) {
    T* buffer = (T*) ((size_t) workspace + offset);
    size_t size = elements * sizeof(T);
    offset += size;
    return buffer;
}

pluginStatus_t YOLOv8PoseLayerLauncher(
    YOLOv8PoseLayerParameters param,
    const void* reg1Input, const void* reg2Input, const void* reg3Input,
    const void* cls1Input, const void* cls2Input, const void* cls3Input,
    const void* ps1Input, const void* ps2Input, const void* ps3Input,
    void* numDetectionsOutput, void* nmsClassesOutput, void* nmsScoresOutput, 
    void* nmsBoxesOutput, void* nmsKeyPointsOutput, void* workspace, hipStream_t stream
) {
    hipMemsetAsync(numDetectionsOutput, 0, sizeof(int) * param.batchSize, stream);
    hipMemsetAsync(nmsClassesOutput, 0, sizeof(int) * param.batchSize * param.numOutputBoxes, stream);
    hipMemsetAsync(nmsScoresOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes, stream);
    hipMemsetAsync(nmsBoxesOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes * 4, stream);
    hipMemsetAsync(nmsKeyPointsOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes * param.numKeypoints * 3, stream);

    // Counters Workspace
    size_t workspaceOffset = 0;
    int rects_element = (4 + 1) * param.batchSize * param.numAnchors;
    int* outputRects = YOLOv8PoseLayerWorkspace<int>(workspace, workspaceOffset, rects_element);

    int socres_element = param.batchSize * param.numAnchors;
    float* outputScores = YOLOv8PoseLayerWorkspace<float>(workspace, workspaceOffset, socres_element);

    int count_element = param.batchSize;
    int* outputCount = YOLOv8PoseLayerWorkspace<int>(workspace, workspaceOffset, count_element);
    hipMemsetAsync(outputCount, 0, sizeof(int) * param.batchSize, stream);

    int keep_element = param.batchSize * param.numAnchors;
    int* outputKeep = YOLOv8PoseLayerWorkspace<int>(workspace, workspaceOffset, keep_element);
    hipMemsetAsync(outputKeep, -1, sizeof(int) * param.numAnchors * param.batchSize, stream);

    int threadSize = 256;
    dim3 block(threadSize, 1);
    dim3 grid(param.batchSize, (param.numAnchors + threadSize - 1) / threadSize);

    YOLOv8PoseLayerNMS(
        param,
        (const float *)reg1Input, (const float *)reg2Input, (const float *)reg3Input, 
        (const float *)cls1Input, (const float *)cls2Input, (const float *)cls3Input, 
        (const float *)ps1Input, (const float *)ps2Input, (const float *)ps3Input,
        outputRects, outputScores, outputCount, outputKeep,
        (int *)numDetectionsOutput, (int *)nmsClassesOutput, 
        (float *)nmsScoresOutput, (float *)nmsBoxesOutput, 
        (float *)nmsKeyPointsOutput
    );

    hipError_t status = hipGetLastError();
    CSC(status, STATUS_FAILURE);

    return STATUS_SUCCESS;
}

pluginStatus_t YOLOv8PoseLayerInference(
    YOLOv8PoseLayerParameters param,
    const void* reg1Input, const void* reg2Input, const void* reg3Input,
    const void* cls1Input, const void* cls2Input, const void* cls3Input,
    const void* ps1Input, const void* ps2Input, const void* ps3Input,
    void* numDetectionsOutput, void* nmsClassesOutput, void* nmsScoresOutput, 
    void* nmsBoxesOutput, void* nmsKeyPointsOutput, void* workspace, hipStream_t stream
) {
    return YOLOv8PoseLayerLauncher(
        param,
        reg1Input, reg2Input, reg3Input,
        cls1Input, cls2Input, cls3Input,
        ps1Input, ps2Input, ps3Input,
        numDetectionsOutput, nmsClassesOutput, nmsScoresOutput,
        nmsBoxesOutput, nmsKeyPointsOutput, workspace, stream 
    );
}