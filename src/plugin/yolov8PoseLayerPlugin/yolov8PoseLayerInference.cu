#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "yolov8PoseLayerParameters.h"

#include <stdio.h>
__device__ 
float sigmoid_gpu(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__device__
float iou_gpu(
    const float xmin_A, const float ymin_A, const float xmax_A, const float ymax_A,
    const float xmin_B, const float ymin_B, const float xmax_B, const float ymax_B
) {
    float xmin = fmaxf(xmin_A, xmin_B);
    float ymin = fmaxf(ymin_A, ymin_B);
    float xmax = fminf(xmax_A, xmax_B);
    float ymax = fminf(ymax_A, ymax_B);
    float iw = fmaxf(0.0f, xmax - xmin);
    float ih = fmaxf(0.0f, ymax - ymin);
    float inter = iw * ih;
    float area1 = (xmax_A- xmin_A) * (ymax_A - ymin_A);
    float area2 = (xmax_B - xmin_B) * (ymax_B - ymin_B);
    return inter / (area1 + area2 - inter);
}

__global__
void YOLOv8PoseLayerNMS(
    YOLOv8PoseLayerParameters param,
    const float*  reg1Data, const float*  reg2Data, const float*  reg3Data, 
    const float*  cls1Data, const float*  cls2Data, const float*  cls3Data, 
    const float*  ps1Data, const float*  ps2Data, const float*  ps3Data,
    float* outputRects, int* outputClasses, int* outputCount, int* outputKeep,
    int* __restrict__ numDetectionsOutput, int* __restrict__ nmsClassesOutput, 
    float* __restrict__ nmsScoresOutput, float* __restrict__ nmsBoxesOutput, 
    float* __restrict__ nmsKeyPointsOutput
) {
    int imageIdx = blockIdx.x;
    int anchorIdx = blockIdx.y * blockDim.x + threadIdx.x;
    if (imageIdx >= param.batchSize) return;
    if (anchorIdx >= param.numAnchors) return;

    int head_idx = 0;
    int base_tid = anchorIdx % param.numAnchors;

    if (base_tid < param.headStart)
        head_idx = 0;
    else if (base_tid < param.headEnd)
        head_idx = 1;
    else
        head_idx = 2;

    int local_start = (head_idx == 1) * param.headStart + (head_idx == 2) * param.headEnd;
    
    int stride = param.minStride << head_idx;

    int h = param.inputHeight / stride;
    int w = param.inputWidth / stride;
    int local_idx = base_tid - local_start;
    int row = local_idx / w;
    int col = local_idx % w;
    
    const float* reg = (head_idx == 0) ? reg1Data + imageIdx * param.reg1Size : ((head_idx == 1) ? reg2Data + imageIdx * param.reg2Size: reg3Data + imageIdx * param.reg3Size);
    const float* cls = (head_idx == 0) ? cls1Data + imageIdx * param.cls1Size: ((head_idx == 1) ? cls2Data + imageIdx * param.cls2Size : cls3Data + imageIdx * param.cls3Size);
    const float* ps  = (head_idx == 0) ? ps1Data + imageIdx * param.ps1Size : ((head_idx == 1) ? ps2Data + imageIdx * param.ps2Size : ps3Data + imageIdx * param.ps3Size);

    float cx = float(col + 0.5);
    float cy = float(row + 0.5);

    float cls_max = -1;
    int cls_index = -1;
    for (int cl = 0; cl < param.numClasses; cl++) {
        float cls_val = sigmoid_gpu(cls[cl * h * w + row * w + col]);
        if (cls_val > cls_max) {
            cls_max = cls_val;
            cls_index = cl;
        }
    }

    if (cls_max < param.scoreThreshold) return;

    float dx1 = reg[0 * h * w + row * w + col];
    float dy1 = reg[1 * h * w + row * w + col];
    float dx2 = reg[2 * h * w + row * w + col];
    float dy2 = reg[3 * h * w + row * w + col];

    float xmin = (cx - dx1) * stride;
    float ymin = (cy - dy1) * stride;
    float xmax = (cx + dx2) * stride;
    float ymax = (cy + dy2) * stride;
    
    xmin = fmaxf(0.0f, xmin);
    ymin = fmaxf(0.0f, ymin);
    xmax = fminf(param.inputWidth, xmax);
    ymax = fminf(param.inputHeight, ymax);

    int batch_offset = imageIdx * param.numAnchors;

    int id = atomicAdd(&outputCount[imageIdx], 1);

    int index_i = id + batch_offset;
    outputKeep[index_i] = 1;

    float* index_i_rect = outputRects + index_i * (4 + 1);
    int* index_i_class = outputClasses + index_i;
    index_i_rect[0] = xmin;
    index_i_rect[1] = ymin;
    index_i_rect[2] = xmax;
    index_i_rect[3] = ymax;
    index_i_rect[4] = cls_max;
    index_i_class[0] = cls_index;

    __syncthreads();

    for (int j = 0; j < outputCount[imageIdx]; j++) {
        int index_j = j + batch_offset;

        float* index_j_rect = outputRects + index_j * (4 + 1);
        int* index_j_class = outputClasses + index_j;

        if (index_i == index_j || outputKeep[index_j] == 0) continue;
        if (index_i_class[0] == index_j_class[0] && index_i_rect[4] < index_j_rect[4]) {
            float iou = iou_gpu(
                index_i_rect[0], index_i_rect[1], index_i_rect[2], index_i_rect[3], 
                index_j_rect[0], index_j_rect[1], index_j_rect[2], index_j_rect[3]
            );
            if (iou > param.iouThreshold) {
                outputKeep[index_i] = 0;
            }
        }
    }

    __syncthreads();

    if (outputKeep[index_i] == 1) {
        // NumDetections
        int kid = atomicAdd(&numDetectionsOutput[imageIdx], 1);

        // DetectionClasses
        nmsClassesOutput[kid + imageIdx * param.numOutputBoxes] = index_i_class[0];

        // DetectionScores
        nmsScoresOutput[kid + imageIdx * param.numOutputBoxes] = index_i_rect[4];

        // DetectionBoxes
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 0] = index_i_rect[0];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 1] = index_i_rect[1];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 2] = index_i_rect[2];
        nmsBoxesOutput[(kid + imageIdx * param.numOutputBoxes) * 4 + 3] = index_i_rect[3];

        // DetectionKeyPoints
        for (int k = 0; k < param.numKeypoints; k++) {
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 0] = (ps[(k * 3 + 0) * h * w + row * w + col] * 2 + (cx - 0.5f)) * stride;
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 1] = (ps[(k * 3 + 1) * h * w + row * w + col] * 2 + (cy - 0.5f)) * stride;
            nmsKeyPointsOutput[(k + imageIdx * param.numOutputBoxes) * param.numKeypoints * 3 + kid * 3 + 2] = sigmoid_gpu(ps[(k * 3 + 2) * h * w + row * w + col]);
        }
    }
}

template <typename T>
T* YOLOv8PoseLayerWorkspace(void* workspace, size_t& offset, size_t elements) {
    T* buffer = (T*) ((size_t) workspace + offset);
    size_t size = elements * sizeof(T);
    offset += size;
    return buffer;
}

pluginStatus_t YOLOv8PoseLayerLauncher(
    YOLOv8PoseLayerParameters param,
    const void* reg1Input, const void* reg2Input, const void* reg3Input,
    const void* cls1Input, const void* cls2Input, const void* cls3Input,
    const void* ps1Input, const void* ps2Input, const void* ps3Input,
    void* numDetectionsOutput, void* nmsClassesOutput, void* nmsScoresOutput, 
    void* nmsBoxesOutput, void* nmsKeyPointsOutput, void* workspace, hipStream_t stream
) {
    hipMemsetAsync(numDetectionsOutput, 0, sizeof(int) * param.batchSize, stream);
    hipMemsetAsync(nmsClassesOutput, 0, sizeof(int) * param.batchSize * param.numOutputBoxes, stream);
    hipMemsetAsync(nmsScoresOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes, stream);
    hipMemsetAsync(nmsBoxesOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes * 4, stream);
    hipMemsetAsync(nmsKeyPointsOutput, 0, sizeof(float) * param.batchSize * param.numOutputBoxes * param.numKeypoints * 3, stream);

    // Counters Workspace
    size_t workspaceOffset = 0;
    int rects_element = (4 + 1) * param.batchSize * param.numAnchors;
    float* outputRects = YOLOv8PoseLayerWorkspace<float>(workspace, workspaceOffset, rects_element);

    int classes_element = param.batchSize * param.numAnchors;
    int* outputClasses = YOLOv8PoseLayerWorkspace<int>(workspace, workspaceOffset, classes_element);

    int count_element = param.batchSize;
    int* outputCount = YOLOv8PoseLayerWorkspace<int>(workspace, workspaceOffset, count_element);
    hipMemsetAsync(outputCount, 0, sizeof(int) * param.batchSize, stream);

    int keep_element = param.batchSize * param.numAnchors;
    int* outputKeep = YOLOv8PoseLayerWorkspace<int>(workspace, workspaceOffset, keep_element);
    hipMemsetAsync(outputKeep, -1, sizeof(int) * param.numAnchors * param.batchSize, stream);

    int threadSize = 256;
    dim3 block(threadSize, 1);
    dim3 grid(param.batchSize, (param.numAnchors + threadSize - 1) / threadSize);

    YOLOv8PoseLayerNMS<<<grid, block, 0, stream>>>(
        param,
        (const float *)reg1Input, (const float *)reg2Input, (const float *)reg3Input, 
        (const float *)cls1Input, (const float *)cls2Input, (const float *)cls3Input, 
        (const float *)ps1Input, (const float *)ps2Input, (const float *)ps3Input,
        outputRects, outputClasses, outputCount, outputKeep,
        (int *)numDetectionsOutput, (int *)nmsClassesOutput, 
        (float *)nmsScoresOutput, (float *)nmsBoxesOutput, 
        (float *)nmsKeyPointsOutput
    );

    hipError_t status = hipGetLastError();
    CSC(status, STATUS_FAILURE);

    return STATUS_SUCCESS;
}

pluginStatus_t YOLOv8PoseLayerInference(
    YOLOv8PoseLayerParameters param,
    const void* reg1Input, const void* reg2Input, const void* reg3Input,
    const void* cls1Input, const void* cls2Input, const void* cls3Input,
    const void* ps1Input, const void* ps2Input, const void* ps3Input,
    void* numDetectionsOutput, void* nmsClassesOutput, void* nmsScoresOutput, 
    void* nmsBoxesOutput, void* nmsKeyPointsOutput, void* workspace, hipStream_t stream
) {
    return YOLOv8PoseLayerLauncher(
        param,
        reg1Input, reg2Input, reg3Input,
        cls1Input, cls2Input, cls3Input,
        ps1Input, ps2Input, ps3Input,
        numDetectionsOutput, nmsClassesOutput, nmsScoresOutput,
        nmsBoxesOutput, nmsKeyPointsOutput, workspace, stream 
    );
}